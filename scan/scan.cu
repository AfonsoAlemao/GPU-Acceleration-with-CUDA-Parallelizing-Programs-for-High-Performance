#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
if (code != hipSuccess) {
fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file, line);
if (abort) exit(code);
}
}
#else
#define cudaCheckError(ans) ans
#endif

// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}


// This is the CUDA "kernel" function that is run on the GPU.  You
// know this because it is marked as a __global__ function.
__global__ void
upsweepPhaseKernel(int twod1, int twod, int* result, int N) {

    // compute overall thread index from position of thread in current
    // block, and given the block we are in (in this example only a 1D
    // calculation is needed so the code only looks at the .x terms of
    // blockDim and threadIdx.
    int index = (blockIdx.x * blockDim.x + threadIdx.x);
    if (index < N/twod1){
        index *= twod1;
        if (index + twod1 - 1 < N) {
            result[index + twod1 - 1] = result[index + twod - 1] + result[index + twod1 - 1];
        }
    }

}

// This is the CUDA "kernel" function that is run on the GPU.  You
// know this because it is marked as a __global__ function.
__global__ void
downsweepPhaseKernel(int twod1, int twod, int* result, int N, int nextPow2var) {

    // compute overall thread index from position of thread in current
    // block, and given the block we are in (in this example only a 1D
    // calculation is needed so the code only looks at the .x terms of
    // blockDim and threadIdx.
    int index = (blockIdx.x * blockDim.x + threadIdx.x);
    if (index < nextPow2var/twod1){
        index *= twod1;
        if (index + twod1 - 1 < nextPow2var) {
            int aux = result[index + twod1 - 1];
            if (index + twod - 1 < N) {
                int tmp = result[index + twod - 1];
                result[index + twod1 - 1] = tmp + aux;
            }
            result[index + twod - 1] = aux;
        }
    }
}


// This is the CUDA "kernel" function that is run on the GPU.  You
// know this because it is marked as a __global__ function.
__global__ void
initializeResultKernel(int* input, int* result, int N, int nextPow2N) {

    // compute overall thread index from position of thread in current
    // block, and given the block we are in (in this example only a 1D
    // calculation is needed so the code only looks at the .x terms of
    // blockDim and threadIdx.
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N) {
        result[index] = input[index];
    }
    else if (index < nextPow2N) {
        result[index] = 0;
    }
}

__global__ void
putZeroInEnd(int* result, int N) {
    result[N - 1] = 0;
}


// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel segmented scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
void exclusive_scan(int* input, int N, int* result)
{

    // STUDENTS TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.
    
    int nextPow2var = nextPow2(N);
    const int blocks = (nextPow2var + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    initializeResultKernel<<<blocks, THREADS_PER_BLOCK>>>(input, result, N, nextPow2var);
    cudaCheckError(hipDeviceSynchronize());
    
    // Testing
    /* int* resultt = (int*)malloc(N*sizeof(int));
    hipMemcpy(resultt, result, N * sizeof(int), hipMemcpyDeviceToHost);
    printf("Initially\n");
    for (int i = 0; i < nextPow2(N); i++) {
        printf("%d\n", resultt[i]);
    }
    printf("\n"); */

    // upsweep phase
    for (int twod = 1; twod < nextPow2var / 2; twod *= 2) {
        int twod1 = twod*2;
        int num_block_iter = ((nextPow2var/twod1) + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        int threads_per_block = THREADS_PER_BLOCK;
        if (num_block_iter == 1) {
            threads_per_block = (nextPow2var/twod1);
        }
        upsweepPhaseKernel<<<num_block_iter, threads_per_block>>>(twod1, twod, result, nextPow2var);
        cudaCheckError(hipDeviceSynchronize());

        // Testing
        /* hipMemcpy(resultt, result, N * sizeof(int), hipMemcpyDeviceToHost);
        printf("Iteration %d \n", twod);
        for (int i = 0; i < nextPow2(N); i++) {
            printf("A[%d]=%d\n", i, resultt[i]);
        }
        printf("\n"); */
    }
    
    putZeroInEnd<<<1, 1>>>(result, nextPow2var);
    cudaCheckError(hipDeviceSynchronize());
     // Testing
    /* hipMemcpy(resultt, result, N * sizeof(int), hipMemcpyDeviceToHost);
    printf("End\n");
    for (int i = 0; i < nextPow2(N); i++) {
        printf("A[%d]=%d\n", i, resultt[i]);
    }
    printf("\n");*/

    // downsweep phase
    for (int twod = nextPow2var / 2; twod >= 1; twod /= 2) {
        int twod1 = twod * 2;
        int num_block_iter = ((nextPow2var/twod1) + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        int threads_per_block = THREADS_PER_BLOCK;
        if (num_block_iter == 1) {
            threads_per_block = (nextPow2var/twod1);
        }
        downsweepPhaseKernel<<<num_block_iter, threads_per_block>>>(twod1, twod, result, N, nextPow2var);
        cudaCheckError(hipDeviceSynchronize());
    }

    // Testing
    /* hipMemcpy(resultt, result, N * sizeof(int), hipMemcpyDeviceToHost);
    printf("End\n");
    for (int i = 0; i < nextPow2(N); i++) {
        printf("A[%d]=%d\n", i, resultt[i]);
    }
    printf("\n"); */

}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of segmented scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_result);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


__global__ void
isEqualToNext(int N, int* aux, int* input) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int support[THREADS_PER_BLOCK + 1];
    
    support[threadIdx.x] = input[index];
    if (threadIdx.x < 1) {
        support[THREADS_PER_BLOCK + threadIdx.x] = input[index + THREADS_PER_BLOCK];
    }

    __syncthreads();

    if (index < N - 1) {
        if (support[threadIdx.x] == support[threadIdx.x + 1]) {
            aux[index] = 1;
        }
        else {
            aux[index] = 0;
        }
    }
    
}

__global__ void
getFindRepeats(int N, int* resultarray, int* device_output) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int auxiliar = 0;

    __shared__ int support[THREADS_PER_BLOCK + 1];
    
    support[threadIdx.x] = resultarray[index];
    if (threadIdx.x < 1) {
        support[THREADS_PER_BLOCK + threadIdx.x] = resultarray[index + THREADS_PER_BLOCK];
    }
    __syncthreads();

    if (index < N - 1) {
        auxiliar = support[threadIdx.x];
        if (auxiliar != support[threadIdx.x + 1]) {
            device_output[auxiliar] = index;
        }
    }

}

__global__ void
switchlast_first(int length, int* device_input) {    
    device_input[0] = device_input[length - 1];
}

// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // STUDENTS TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.

    const int blocks = (length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int resultarray;

    // Testing
    /* int* resultt = (int*)malloc(nextPow2var*sizeof(int));
    hipMemcpy(resultt, device_input, nextPow2var * sizeof(int), hipMemcpyDeviceToHost);
    printf("Initialy\n");
    for (int i = 0; i < nextPow2var; i++) {
        printf("A[%d]=%d\n", i, resultt[i]);
    }
    printf("\n"); */

    isEqualToNext<<<blocks, THREADS_PER_BLOCK>>>(length, device_output, device_input);
    cudaCheckError(hipDeviceSynchronize());
    // Testing
    /* hipMemcpy(resultt, device_output, nextPow2var * sizeof(int), hipMemcpyDeviceToHost);
    printf("IsEqualToNext\n");
    for (int i = 0; i < nextPow2var; i++) {
        printf("A[%d]=%d\n", i, resultt[i]);
    }
    printf("\n"); */ 

    cudaScan(device_output, device_output + length, device_input);

    /* for (int i = 0; i < nextPow2var; i++){
        printf("Ressultarray: %d\n", resultarray[i]);
    }
    printf("\n"); */

    getFindRepeats<<<blocks, THREADS_PER_BLOCK>>>(length, device_input, device_output);
    cudaCheckError(hipDeviceSynchronize());
     // Testing
    /* hipMemcpy(resultt, device_output, number_pairs * sizeof(int), hipMemcpyDeviceToHost);
    printf("Device output\n");
    for (int i = 0; i < resultarray[nextPow2var - 1]; i++) {
        printf("A[%d]=%d\n", i, resultt[i]);
    }
    printf("\n"); */ 

    switchlast_first<<<1, 1>>>(length, device_input);
    cudaCheckError(hipDeviceSynchronize());
    hipMemcpy(&resultarray, device_input, sizeof(int), hipMemcpyDeviceToHost);
    return resultarray; 
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}