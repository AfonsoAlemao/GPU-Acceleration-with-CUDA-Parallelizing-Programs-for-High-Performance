#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"


// return GB/sec
float GBPerSec(int bytes, float sec) {
  return static_cast<float>(bytes) / (1024. * 1024. * 1024.) / sec;
}


// This is the CUDA "kernel" function that is run on the GPU.  You
// know this because it is marked as a __global__ function.
__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall thread index from position of thread in current
    // block, and given the block we are in (in this example only a 1D
    // calculation is needed so the code only looks at the .x terms of
    // blockDim and threadIdx.
    int index = blockIdx.x * blockDim.x + threadIdx.x;


    // this check is necessary to make the code work for values of N
    // that are not a multiple of the thread block size (blockDim.x)
    if (index < N)
       result[index] = alpha * x[index] + y[index];
}


// saxpyCuda --
//
// This function is regular C code running on the CPU.  It allocates
// memory on the GPU using CUDA API functions, uses CUDA API functions
// to transfer data from the CPU's memory address space to GPU memory
// address space, and launches the CUDA kernel function on the GPU.
void saxpyCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {

    // must read both input arrays (xarray and yarray) and write to
    // output array (resultarray)
    int totalBytes = sizeof(float) * 3 * N;

    // compute number of blocks and threads per block.  In this
    // application we've hardcoded thread blocks to contain 512 CUDA
    // threads.
    const int threadsPerBlock = 512;

    // Notice the round up here.  The code needs to compute the number
    // of threads blocks needed such that there is one thread per
    // element of the arrays.  This code is written to work for values
    // of N that are not multiples of threadPerBlock.
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    // These are pointers that will be pointers to memory allocated
    // *one the GPU*.  You should allocate these pointers via
    // hipMalloc.  You can access the resulting buffers from CUDA
    // device kernel code (see the kernel function saxpy_kernel()
    // above) but you cannot access the contents these buffers from
    // this thread. CPU threads cannot issue loads and stores from GPU
    // memory!
    float* device_x;
    float* device_y;
    float* device_result;

    //
    // STUDENTS TODO: allocate device memory buffers on the GPU using hipMalloc.
    //
    // We highly recommend taking a look at NVIDIA's
    // tutorial, which clearly walks you through the few lines of code
    // you need to write for this part of the assignment:
    //
    // https://devblogs.nvidia.com/easy-introduction-cuda-c-and-c/
    //
    hipMalloc(&device_x, N*sizeof(float)); 
    hipMalloc(&device_y, N*sizeof(float));
    hipMalloc(&device_result, N*sizeof(float)); 
        
    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    //
    // STUDENTS TODO: copy input arrays to the GPU using hipMemcpy
    //
    hipMemcpy(device_x, xarray, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_y, yarray, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_result, resultarray, N*sizeof(float), hipMemcpyHostToDevice);

   
    double startTime2 = CycleTimer::currentSeconds();
    // run CUDA kernel. (notice the <<< >>> brackets indicating a CUDA
    // kernel launch) Execution on the GPU occurs here.
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);
    cudaCheckError(hipDeviceSynchronize()); // error is printed on this line
    double endTime2 = CycleTimer::currentSeconds(); 

    //
    // STUDENTS TODO: copy result from GPU back to CPU using hipMemcpy
    //
    hipMemcpy(xarray, device_x, N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(yarray, device_y, N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(resultarray, device_result, N*sizeof(float), hipMemcpyDeviceToHost);
    
    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n",
		errCode, hipGetErrorString(errCode));
    }

    // double overallDuration = endTime - startTime;
    // printf("Effective BW by CUDA saxpy: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, GBPerSec(totalBytes, overallDuration));

    double overallDuration2 = endTime2 - startTime2;
    printf("Time taken to run the kernel: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration2, GBPerSec(totalBytes, overallDuration2));

    //
    // STUDENTS TODO: free memory buffers on the GPU using hipFree
    //
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);
}

void printCudaInfo() {

    // print out stats about the GPU in the machine.  Useful if
    // students want to know what GPU they are running on.

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
if (code != hipSuccess) {
fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file, line);
if (abort) exit(code);
}
}
#else
#define cudaCheckError(ans) ans
#endif